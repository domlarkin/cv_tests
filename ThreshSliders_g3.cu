#include "hip/hip_runtime.h"
/* OpenCV3: Thresholding using GPU */
#include <iostream>
#include <string.h>
#include <time.h>
#include <ctime>
#include <chrono>

//#include "opencv2/opencv.hpp"
//#include "opencv2/imgcodecs.hpp"
//#include "opencv2/core/cuda.hpp"
//#include "opencv2/cudaarithm.hpp"

#include "opencv2/core.hpp"
#include <opencv2/core/utility.hpp>
#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/cudaimgproc.hpp"
#include "opencv2/cudafilters.hpp" // used for gaussian
#include "opencv2/cudaarithm.hpp" // Used for threshold
#include "opencv2/core/matx.hpp" // Used for vec4i

#include <fstream>
#include <sstream>

using namespace cv;

int threshold_value = 225;
int threshold_type = 3;
int erosion_size = 1;
int bkSize =5;
int ekSize =3;
int h_rho = 1; // Distance resolution of the accumulator in pixels (hough transform)
int h_theta = 180; // Angle resolution of the accumulator in radians (hough transform)
int h_minLineLen = 150; // Line segments shorter than that are rejected (hough transform)
int h_maxLineGap = 25; // Maximum allowed gap between points on the same line to link them (hough transform)

//std::vector<cv::Vec4i> lines;
cv::cuda::GpuMat d_lines;

int const max_value = 255;
int const max_type = 4;
int const max_BINARY_value = 255;
cv::cuda::GpuMat src, src_gray, src_blur, src_erode, src_canny, dst;
cv::Mat src_host; 
const char* window_name = "Threshold Demo";
const char* trackbar_type = "Type: \n 0: Binary \n 1: Binary Inverted \n 2: Truncate \n 3: To Zero \n 4: To Zero Inverted";
const char* trackbar_value = "Value";
void Threshold_Demo( int, void* );

void log2file( const std::string &text )
{
    std::ofstream log_file(
        "log_file_cuda.csv", std::ios_base::out | std::ios_base::app );
    log_file << text;
}
bool isLogging=false;

int main( int, char** argv )
{
    namedWindow( window_name, WINDOW_AUTOSIZE );
    createTrackbar( trackbar_type, window_name, &threshold_type, max_type);
    createTrackbar( trackbar_value, window_name, &threshold_value, max_value);
    createTrackbar( "Blur kSize", window_name, &bkSize, 9);
    createTrackbar( "Erode kSize", window_name, &ekSize, 9);
    createTrackbar( "h_rho", window_name, &h_rho, 10);
    createTrackbar( "h_theta", window_name, &h_theta, 180);
    createTrackbar( "h_minLineLen", window_name, &h_minLineLen, 400);
    createTrackbar( "h_maxLineGap", window_name, &h_maxLineGap, 100);


    VideoCapture capture("leftcam_15.avi");
    if(!capture.isOpened()){
        std::cout<<"cannot read video!\n";
        return -1;
    }
    std::ostringstream colHeaders;
    colHeaders << "top,aftblur,aftgray,aftthresh,afterode,aftcanny,afthough,aftdrawing,"<<CLOCKS_PER_SEC<<"\n" ;
    if (isLogging) log2file(colHeaders.str());
    bool pause = false;
    while(true)
    {
        if(!pause){
            if(!capture.read(src_host)){
                break;
            }
        }
        Threshold_Demo(0,0);
        int c = waitKey(20);
        if((char)c == 27) { 
            break;
        }
        else if(c == 'p'){
            pause = !pause;
        } 
    }
}

void Threshold_Demo( int, void* )
{ 
  /* 0: Binary
     1: Binary Inverted
     2: Threshold Truncated
     3: Threshold to Zero
     4: Threshold to Zero Inverted
   */
   
   /*
     *  It Uses the following algorithm to find white lines:
     *     1. blur the image
     *     2. turn image into grayscale
     *     3. run it through a threshold filter using THRESH_TO_ZERO mode
     *     4. run it through an erosion filter
     *     5. run it through a Canny edge detector
     *     6. finally, take this processed image and find the lines using   
   */
   // Blur the image
    std::ostringstream outStream;
    src.upload(src_host); 
    if (isLogging) outStream << clock() << ","; // TIMER: top
    
    // ===== Blur the image CUDA
    cv::Ptr<cv::cuda::Filter> filter = cv::cuda::createGaussianFilter(src.type(), src_blur.type(), cv::Size((bkSize*2)+1, (bkSize*2)+1), 0.0);
    filter->apply(src, src_blur);
    if (isLogging) outStream << clock() << ","; // TIMER: aftblur 
   
    // ===== GrayScale the image CUDA    
    cv::cuda::cvtColor( src_blur, src_gray, COLOR_RGB2GRAY );
    if (isLogging) outStream << clock() << ","; // TIMER: aftgray 
    
    // ===== Threshhold the image CUDA
    cv::cuda::threshold( src_gray, src_erode, threshold_value, max_BINARY_value,threshold_type );
    if (isLogging) outStream << clock() << ","; // TIMER: aftthresh 

    // ===== Erode the image CUDA
    cv::Mat element = getStructuringElement(cv::MORPH_ELLIPSE, cv::Size((ekSize*2)+1, (ekSize*2)+1),
                                            cv::Point(-1, -1));
    Ptr<cuda::Filter> erodeFilter = cv::cuda::createMorphologyFilter(MORPH_ERODE, src_erode.type(), element);
    erodeFilter->apply(src_erode, src_canny);
    if (isLogging) outStream << clock() << ","; // TIMER: afterode   

    // ===== Canny edge detection CUDA
    cv::Ptr<cv::cuda::CannyEdgeDetector> canny = cv::cuda::createCannyEdgeDetector(50, 250, 3);
    canny->detect(src_canny, dst);
    if (isLogging) outStream << clock() << ","; // TIMER: aftcanny 

    // ===== Find the Hough lines CUDA
    //cv::Ptr<cv::cuda::HoughSegmentDetector> hough = cv::cuda::createHoughSegmentDetector(1.0f, (float) (CV_PI / 180.0f), 50, 5);
    cv::Ptr<cv::cuda::HoughSegmentDetector> hough = cv::cuda::createHoughSegmentDetector(h_rho, (CV_PI / h_theta), h_minLineLen, h_maxLineGap);
    hough->detect(src_canny, d_lines);
    if (isLogging) outStream << clock() << ","; // TIMER: afthough
    std::vector<cv::Vec4i> lines_gpu;
    if (!d_lines.empty())
    {
        lines_gpu.resize(d_lines.cols);
        Mat h_lines(1, d_lines.cols, CV_32SC4, &lines_gpu[0]);
        d_lines.download(h_lines);
    }
    Mat dst_host;
    src.download(dst_host);
    for (size_t i = 0; i < lines_gpu.size(); ++i)
    {
        Vec4i l = lines_gpu[i];
        line(dst_host, Point(l[0], l[1]), Point(l[2], l[3]), Scalar(0, 0, 255), 3, LINE_AA);
    }
    
    if (isLogging) outStream << clock() << "\n"; // TIMER: aftdrawing
    if (isLogging) log2file(outStream.str());
    imshow( window_name, dst_host );
}
