#include "hip/hip_runtime.h"
/* OpenCV3: Thresholding using GPU */
#include <iostream>
#include <string.h>
#include <time.h>
#include <ctime>
#include <chrono>
#include "opencv2/opencv.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/core/cuda.hpp"
#include "opencv2/cudaarithm.hpp"
#include <fstream>
#include <sstream>
using namespace cv;

void log2file( const std::string &text )
{
    std::ofstream log_file(
        "log_fileg3.csv", std::ios_base::out | std::ios_base::app );
    log_file << text;
}

bool isLogging=true;

int main (int argc, char* argv[])
{
    try
    {
        for(int i = 0; i < 1000; i++){
            std::ostringstream outStream;            
            cv::Mat src_host = cv::imread("file.png", IMREAD_GRAYSCALE);
            if (isLogging) outStream << clock() << ","; // TIMER: top
            cv::cuda::GpuMat dst, src;
            src.upload(src_host);

            if (isLogging) outStream << clock() << ","; // TIMER: top
            cv::cuda::threshold(src, dst, 128.0, 255.0, THRESH_BINARY);

            if (isLogging) outStream << clock() << ","; // TIMER: top
            cv::Mat result_host;
            dst.download(result_host);
            if (isLogging) outStream << clock() << "\n"; // TIMER: bottom
            if (isLogging) log2file(outStream.str());
            //cv::imshow("Result", result_host);
            //cv::waitKey();
        }
    }
    catch(const cv::Exception& ex)
    {
        std::cout << "Error: " << ex.what() << std::endl;
    }
    return 0;
}
